
#include <hip/hip_runtime.h>
#include <stdio.h>

#define TILE_SIZE 16

__global__ void mysgemm(int m, int n, int k, const float *A, const float *B, float* C) {

    /********************************************************************
     *
     * Compute C = A x B
     *   where A is a (m x k) matrix
     *   where B is a (k x n) matrix
     *   where C is a (m x n) matrix
     *
     * Use shared memory for tiling
     *
     ********************************************************************/

    /*************************************************************************/
    // INSERT KERNEL CODE HERE
    //declare shared memory/thread ints
    __shared__ float ds_M[TILE_SIZE][TILE_SIZE];
    __shared__ float ds_N[TILE_SIZE][TILE_SIZE];


    //declare matrix index vars
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int bx = blockIdx.x;
    int by = blockIdx.y;

    ///define row/col
    int Row = by * blockDim.y + ty;
    int Col = bx * blockDim.x + tx;

    float pval = 0.0;
    
    // load data from A,B into shared mem + boundary checking & zero padding
    for(int p = 0; p < (k-1)/TILE_SIZE + 1; p++)
    {
    	if(Row < m && p * TILE_SIZE + tx < k) //load M - within boundary;  note: (a = m x k)
	{
	    ds_M[ty][tx] = A[Row * k + p * TILE_SIZE + tx];
	}
	else // pad 0
	{
	   ds_M[ty][tx] = 0.0;
	}
	if(p*TILE_SIZE + ty < k && Col < n) // load N - within boundary; note: (b = k x n)
	{
	   ds_N[ty][tx] = B[(p*TILE_SIZE + ty) * n + Col];
	}
	else // pad 0
	{
	    ds_N[ty][tx] = 0.0;
	}

        __syncthreads();        

        if(Row < m && Col < n)
        {
            for(int i = 0; i < TILE_SIZE; i++)
            {
                pval += ds_M[ty][i] * ds_N[i][tx];    
            }
        }
	__syncthreads();	
    } //end outer for

    if(Row < m && Col < n)
    {
        C[Row*n+Col] = pval;
    }
    /*************************************************************************/
}

void basicSgemm(int m, int n, int k, const float *A, const float *B, float *C)
{
    // Initialize thread block and kernel grid dimensions ---------------------

    const unsigned int BLOCK_SIZE = TILE_SIZE;
	
    /*************************************************************************/
    //INSERT CODE HERE
    dim3 DimGrid((n-1)/BLOCK_SIZE + 1 , (m-1)/BLOCK_SIZE + 1,1);
    dim3 DimBlock(BLOCK_SIZE, BLOCK_SIZE, 1);
    /*************************************************************************/

    // Invoke CUDA kernel -----------------------------------------------------
    
    /*************************************************************************/
    //INSERT CODE HERE
    mysgemm<<<DimGrid,DimBlock>>>(m,n,k,A,B,C);	
    /***************x*********************************************************/
}

